#include <cstdio>
#include <vector>
#include <string>
#include <queue>
#include <iostream>
#include <sstream>
#include <chrono>
#include "hip/hip_runtime.h"

using namespace std;

#define Dim 512

struct edge_2{
    vector<int> arg1 = {};
    vector<int> arg2 = {};
    int prev;
    int curr;

    void init_diffatomlist(){
        prev = curr = arg1.size();
    }

    void move_diffatomlist(){
        prev = curr;
        curr = arg1.size();
    }
};

void input(edge_2 *al){
    string l,a_name, arg;
    cout << "called input" << endl;
    while(getline(std::cin, l)){
        istringstream stream(l);
        while(getline(stream, a_name, '(')){
            getline(stream, arg, ',');
            int tmp = atoi(arg.c_str());
            al->arg1.push_back(tmp);
            getline(stream, arg, ')');
            tmp = atoi(arg.c_str());
            al->arg2.push_back(atoi(arg.c_str()));
        }
    }
}

void output(edge_2 *al){
    int i;
    for (i = 0; i < al->arg1.size()-2; i++)
    {
        cout << "edge(" << al->arg1[i] << "," << 
                           al->arg2[i] << "), ";
    }
    cout << "edge(" << al->arg1[i] << "," << al->arg2[i] << ")." << endl;
}



__global__ 
void parallel_matching(){
    
}

int main(void){
    struct edge_2 atomlist;
    input(&atomlist);
    atomlist.init_diffatomlist();

    output(&atomlist);
    return 0;
}