#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <string>
#include <queue>
#include <sstream>
#include <chrono>
#include "hip/hip_runtime.h"

using namespace std;

#define Dim 512

/** 
 * - step 1 : 出現する可能性のあるアトムの構造体を定義する？
 * -- 一旦非連結用に作る（引数は全てデータアトム）
 * -- 連結にするにしてもポインタを引数分用意すればいいと思う
 * -- atomlist添え字アクセスや連続したメモリ領域の確保のためにベクタを使う
 * -- cchr と同様に freelist も用意する
 */ 
struct edge_2{
    int arg[2];
    bool live = true;
}

struct atomlist{
    vector<edge_2> atom;
    int prev;
    int curr;
    queue<edge_2> freelist;

    void move_diffatomlist(){
        prev = curr;
        curr = atom.size();
    }

    void del_all(){
        while(!(atom.back().live)){
            atom.pop_back();
        }
        while (!(freelist.empty()))
        {
            int adr = freelist.front();
            freelist.pop();
            if(adr < atom.size()){
                atom[adr] = atom.back();
                atom.pop_back();
            }
        } 
    }
}

void input(atomlist *al){
    string l,a_name, arg;
    edge_2 atom;
    while(getline(std::cin, l)){
        istringstream stream(l);
        while(getline(stream, a_name, '(')){
            getline(stream, arg, ',');
            atom.arg[0] = atoi(arg);
            getline(stream, arg, ')');
            atom.arg[1] = atoi(arg);
            al->atom.push_back()
        }
    }
}

void output(atomlist al){
    for (int i = 0; i < al.atom.size(); i++)
    {
        cout << "edge(" << al.atom[i].arg[0] << "," << 
                           al.atom[i].arg[1] << "), ";
    }
    cout << endl;
}

__global__ 
void parallel_matching(){
    
}